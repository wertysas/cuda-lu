

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "common.h"

#define N 3

using data_type = float;


int main() {
  float hostA[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  float hostC[9];
  int  m=N, n=N, lda=N, ldb=N, ldc=N;
  float alpha=1, beta=0;
  size_t sizeA = sizeof(data_type)*N*N;
  size_t sizeC = sizeA;
  float *deviceA, *deviceC;
  hipMalloc(&deviceA, sizeA);
  hipMalloc(&deviceC, sizeC);
  
  hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, sizeC, hipMemcpyHostToDevice);
    
  hipblasHandle_t* cublasH = nullptr;
  hipblasCreate(cublasH); // note we dont catch error here!
  // hipStream_t stream;
  // hipStreamCreate(&stream);
  // hipblasSetStream(*cublasH, stream); 

  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_T;
  

  hipblasStatus_t err = hipblasSgeam(*cublasH, transa, transb,
                                  m, n,
                                  &alpha,
                                  deviceA, lda,
                                  &beta,
                                  nullptr, ldb,
                                  deviceC, ldc);
    
  

  hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
  
  std::printf("A matrix print:\n");
  print_matrix(hostA, N);
  std::printf("C matrix print:\n");
  print_matrix(hostC, N);
  // hipsolverHandle_t* cusolverHandle;
  // hipsolverStatus_t cusolverStatus = hipsolverDnCreate(cusolverHandle);



}


