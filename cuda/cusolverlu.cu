

#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

#include "common.h"

#define N 3

using data_type = float;


int main() {
  float hostA[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  float hostC[9];
  int  m=N, n=N, lda=N, ldb=N, ldc=N;
  size_t sizeA = sizeof(data_type)*N*N;
  size_t sizeC = sizeA;
  float *deviceA, *deviceC;
  hipMalloc(&deviceA, sizeA);
  hipMalloc(&deviceC, sizeC);
  
  hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, sizeC, hipMemcpyHostToDevice);

  std::printf("A matrix print:\n");
  print_matrix(hostA, N); 

  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle); // note we dont catch error here!
  
  // Matrix transposition using cuBLAS Sgeam see documentation at:
  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-geam
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  float alpha=1, beta=0;
  hipblasStatus_t blasStatus = hipblasSgeam(cublasHandle, transa, transb,
                                  m, n,
                                  &alpha,
                                  deviceA, lda,
                                  &beta,
                                  nullptr, ldb,
                                  deviceC, ldc);
    
  

  hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
  
  std::printf("A matrix print:\n");
  print_matrix(hostA, N);
  std::printf("C matrix print:\n");
  print_matrix(hostC, N);
  
  

  hipsolverHandle_t cusolverHandle;
  hipsolverStatus_t cusolverStatus = hipsolverDnCreate(&cusolverHandle);
  hipsolverDnParams_t cusolverParams;
  hipsolverDnCreateParams(&cusolverParams); // default initialization we don't use advanced options

  // Creating Host and Device Buffers Required by LU solver
  size_t hostBufferSize = 0, deviceBufferSize=0;
  hipsolverDnXgetrf_bufferSize(cusolverHandle, cusolverParams,
    m, n,
    HIP_R_32F, deviceC, ldc,
    HIP_R_32F,
    &deviceBufferSize,
    &hostBufferSize);
  data_type *deviceBuffer, *hostBuffer;
  hipMalloc(&deviceBuffer, sizeof(data_type)*deviceBufferSize);
  hostBuffer = (data_type *) malloc(sizeof(data_type)*deviceBufferSize);


  //LU decomposition using cuSOLVER hipsolverDnXgetrf()
  // see docoumentation at: https://docs.nvidia.com/cuda/cusolver/index.html#cusolverdnxgetrf
  int hostInfo=0;
  int* deviceInfo;
  hipMemcpy(deviceInfo, &hostInfo, sizeof(int), hipMemcpyHostToDevice);
  hipsolverDnXgetrf(cusolverHandle, cusolverParams,
      m, n, HIP_R_32F, deviceC, ldc, nullptr, HIP_R_32F,
      deviceBuffer, deviceBufferSize, hostBuffer, hostBufferSize, deviceInfo);

  hipMemcpy(&hostInfo, deviceInfo, sizeof(int), hipMemcpyDeviceToHost);
  printf("info (should be 0 if LU successful) %d\n", hostInfo);


  hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
  std::printf("LU decomposed C matrix print:\n");
  print_matrix(hostC, N);

  hipblasSgeam(cublasHandle, transa, transb,
    m, n,
    &alpha,
    deviceC, ldc,
    &beta,
    nullptr, ldb,
    deviceA, lda);
    
  hipMemcpy(hostA, deviceA, sizeA, hipMemcpyDeviceToHost);
  std::printf("LU decomposed A matrix print:\n");
  print_matrix(hostA, N);

  // Destroy cuBLAS Handle
  hipblasDestroy(cublasHandle);
  // Destroy cusolverDnParams
  hipsolverDnDestroyParams(cusolverParams);
  // Destroy cuSOLVER Handle
  hipsolverDnDestroy(cusolverHandle);


  hipFree(deviceA);
  hipFree(deviceC);

}
